#ifdef __cplusplus
extern "C" {
#include <stdio.h>
#include "chplgpu.h"

void gpuAllocate(void **data, size_t size) {
  hipError_t err = hipMalloc( data, size);
  if ( hipSuccess != err) {
    fprintf(stderr, "gpuAllocate() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
  printf("gpuAllocate: Allocated %u bytes at %p\n", (unsigned int)size, *data);
}

void gpuFree(void **data) {
  hipError_t err = hipFree( *data);
  if ( hipSuccess != err) {
    fprintf(stderr, "gpuFree() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
}

void copyGPUtoHost(void **dst, void **src, size_t size) {
  printf("** Copying gpu to host : DST = %p SRC = %p SIZE = %u\n", *dst, *src, (unsigned int)size);
  hipError_t err = hipMemcpy( *dst, *src, size, hipMemcpyDeviceToHost);
  if ( hipSuccess != err) {
    fprintf(stderr, "copyGPUtoHost() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
}

void copyHostToGPU(void **dst, void **src, size_t size) {
  printf("** Copying host to gpu : DST = %p SRC = %p SIZE = %u\n", *dst, *src, (unsigned int)size);
  hipError_t err = hipMemcpy( *dst, *src, size, hipMemcpyHostToDevice);
  if ( hipSuccess != err) {
    fprintf(stderr, "copyGPUtoHost() Runtime API error in file <%s>, line %i : %s.\n",
                     __FILE__, __LINE__, hipGetErrorString( err) );
    exit(-1);
  }
}

void chpl_init_accelerator(void) {
  hipSetDevice( 0 ); // init device 0 (this is not clean)
}

}
#endif
